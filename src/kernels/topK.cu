#include "hip/hip_runtime.h"
#include <float.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "src/kernels/topK.h"
#include <hipcub/hipcub.hpp>


template<typename T, int K>
__device__ topK<T, K> reduce_functor(const topK<T, K>& a, const topK<T, K>& b) {
    topK<T, K> res = a;
    for(int i = 0; i < K; i++){
        res.insertHeap(b.val[i], b.id[i]);
    }
    return res;
}

template<typename T, int K, int blockSize, int BlockPerBeam>
__global__ void topK_kernel_round1(const T* probs, const int vocab_size,
                                         int* topK_ids, T* topK_vals)
{
    typedef hipcub::BlockReduce<topK<T, K>, blockSize> blockreduce;
    __shared__ typename blockreduce::TempStorage temp_storage;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int row_id = bid / BlockPerBeam;
    int block_lane = bid % BlockPerBeam;
    topK<T, K> thread_topK;
    thread_topK.init();

    for(int data_id = tid + block_lane * blockSize; data_id < vocab_size; data_id += BlockPerBeam * blockSize){
        int data_offset = data_id + row_id * vocab_size;
        T data = probs[data_offset];
        thread_topK.insertHeap(data, data_offset);
    }

    topK<T, K> block_topK = blockreduce(temp_storage).Reduce(thread_topK, reduce_functor<T, K>);

    if(tid == 0){
        for(int k_offset = 0; k_offset < K; k_offset++) {
            topK_vals[row_id * vocab_size + block_lane * blockSize + k_offset] = block_topK.val[k_offset];
            topK_ids[row_id * vocab_size + block_lane * K + k_offset] = block_topK.id[k_offset];
        }
    }
}

template<typename T, int K, int blockSize, int BlockPerBeam>
__global__ void topK_kernel_round2(const int* topK_ids, const T* topK_vals,
                                    int* final_topK_ids, T* final_topK_vals)
{
    typedef hipcub::BlockReduce<topK<T, K>, blockSize> blockreduce;
    __shared__ typename blockreduce::TempStorage temp_storage;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int row_id = bid;
    topK<T, K> thread_topK;
    thread_topK.init();

    for(int i = tid; i < BlockPerBeam * K; i += blockDim.x) {
        int data_offset = bid * BlockPerBeam * K + i;
        thread_topK.insertHeap(topK_vals[data_offset], topK_ids[i]);
    }

    topK<T, K> block_topK = blockreduce(temp_storage).Reduce(thread_topK, reduce_functor<T, K>);
    if(tid == 0){
        for(int k_offset = 0; k_offset < K; k_offset++) {
            final_topK_vals[bid * K + k_offset] = block_topK.val[k_offset];
            final_topK_ids[bid * K + k_offset] = block_topK.id[k_offset];
        }
    }
}

template <typename T>
void launchTopKforBeamSearch(TensorWrapper<T> *probs,
                             TensorWrapper<int> *topk_ids,
                             TensorWrapper<T> *topk_vals,
                             TensorWrapper<int> *final_topk_ids,
                             TensorWrapper<T> *final_topk_vals)
{
    int bsxbm = probs->shape[0];
    int vocab_size = probs->shape[1];
    constexpr int BlockPerBeam = 8;
    constexpr int beamwidth = 1;
    constexpr int K = 5;

    int topK_val_buf_size = bsxbm * BlockPerBeam * K;
    int topK_ids_buf_size = bsxbm * BlockPerBeam * K;
    int final_topK_val_buf_size = bsxbm * K;

    T* topK_vals_data = topk_vals->data;
    int* topK_ids_data = topk_ids->data;
    T* final_topK_vals_data = final_topk_vals->data;
    int* final_topK_ids_data = final_topk_ids->data;

    int maxBlockNums = 1024; 
    int BlockNums1 = std::min(bsxbm * BlockPerBeam, maxBlockNums);
    int BlockNums2 = std::min(bsxbm, maxBlockNums);
    dim3 grid_round1(BlockNums1);
    dim3 block_round1(256);
    dim3 grid_round2(BlockNums2);
    dim3 block_round2(256);

    topK_kernel_round1<T, K, 256, BlockPerBeam>
                        <<<grid_round1, block_round1>>>(probs->data, vocab_size, topK_ids_data, topK_vals_data);
    topK_kernel_round2<T, K, 256, BlockPerBeam>
                        <<<grid_round2, block_round2>>>(topK_ids_data, topK_vals_data, final_topK_ids_data, final_topK_vals_data);
}


template void launchTopKforBeamSearch(TensorWrapper<float> *probs,
                             TensorWrapper<int> *topk_ids,
                             TensorWrapper<float> *topk_vals,
                             TensorWrapper<int> *final_topk_ids,
                             TensorWrapper<float> *final_topk_vals);