#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h> // For expf
#include "src/kernels/act_kernel.h"
#include "src/utils/cuda_debug_utils.cuh"
#include "src/utils/macro.h"

template<typename T>
__device__ __forceinline__ T silu(const T& in) {
  return (T) (((float) in) / (1.0f + expf((float) -in)));
}

template<typename T>
__global__ void silu_and_mul_kernel(
  T* out,
  const T* input,
  const int intermedia_size) {
  const int batch_idx = blockIdx.x;
  for (int idx = threadIdx.x; idx < intermedia_size; idx += blockDim.x) {
    const int base_idx = batch_idx * 2 * intermedia_size;
    const T x = input[base_idx + idx];
    const T y = input[base_idx + intermedia_size + idx];
    out[batch_idx * intermedia_size + idx] = silu<T>(x) * y;
  }
}

template<typename T>
void launchAct(TensorWrapper<T>* input, TensorWrapper<T>* out) {
    int batch_size = input->shape[0];
    LLM_CHECK(input->shape[1] == 2);
    int intermedia_size = input->shape[2];
    dim3 grid(batch_size);
    dim3 block(256);
    silu_and_mul_kernel<T><<<grid, block>>>(out->data, input->data, intermedia_size);
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(out->data);
#else
#endif
}

template void launchAct(TensorWrapper<float>* input, TensorWrapper<float>* output);